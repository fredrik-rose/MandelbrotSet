#include "hip/hip_runtime.h"
/**
 * \file
 * \brief Mandelbrot set GPU implementation
 */
extern "C"
{
#include <MandelbrotSet/complex.h>
#include <MandelbrotSet/image.h>
#include <MandelbrotSet/mandelbrot.h>
#include <MandelbrotSet/mandelbrot_set.h>
}

#include <assert.h>
#include <stdint.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK_X (16)
#define THREADS_PER_BLOCK_Y (16)
#define NUMBER_OF_BLOCKS (4096)

__global__
static void generate_mandelbrot_set_kernel(
    struct IMG_Image *const image,
    const struct MBROT_Range range,
    const uint8_t max_iterations)
{
    const double x_step = (range.max.real - range.min.real) / image->width;
    const double y_step = (range.max.imag - range.min.imag) / image->height;
    const int x_start = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y_start = (blockIdx.y * blockDim.y + threadIdx.y);
    const int x_stride = blockDim.x * gridDim.x;
    const int y_stride = blockDim.y * gridDim.y;

    for (int y = y_start; y < image->height; y += y_stride)
    {
        for (int x = x_start; x < image->width; x += x_stride)
        {
            const struct CMPLX_Complex c = {
                .real = range.min.real + (x * x_step),
                .imag = range.min.imag + (y * y_step)
            };

            const uint8_t pixel = MBROT_mandelbrot(c, max_iterations);

            IMG_set_pixel(image, x, y, pixel);
        }
    }
}

struct IMG_Image * MBROT_alloc_mandebrot_set_image(
    int width,
    int height)
{
    struct IMG_Image *image;
    const int image_malloc_result = hipMallocManaged(&image, sizeof(*image));
    assert(image_malloc_result == 0);
    assert(image != NULL);

    image->width = width;
    image->height = height;
    const int pixel_malloc_result = hipMallocManaged(&image->pixels, width * height * sizeof(*image->pixels));
    assert(pixel_malloc_result == 0);
    assert(image->pixels != NULL);

    return image;
}

void MBROT_free_mandebrot_set_image(
    struct IMG_Image *const image)
{
    hipFree(image->pixels);
    hipFree(image);
}

void MBROT_generate_mandelbrot_set(
    struct IMG_Image *const image,
    const struct MBROT_Range *const range,
    const uint8_t max_iterations)
{
    assert(image->width > 0);
    assert(image->height > 0);
    assert(range->max.real > range->min.real);
    assert(range->max.imag > range->min.imag);

    const dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    generate_mandelbrot_set_kernel<<<NUMBER_OF_BLOCKS, threadsPerBlock>>>(image, *range, max_iterations);
    hipDeviceSynchronize();
}
